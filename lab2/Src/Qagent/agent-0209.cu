#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 277: GPU Programmming 2021 WINTER
/* Author and Instructer: Cheolhong An
/* Copyright 2019
/* University of California, San Diego
/*************************************************************************/
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define COLS	4
#define ROWS	4

#define NUM_OF_STATES	16
#define NUM_OF_ACTIONS	4

short *d_action;
float *d_epsilon;
hiprandState *d_randstate;
float *d_qtable;

#define gamma	0.9
#define alpha	0.1
// float alpha;

// __device__ float *d_qtable;
short *d_action_buff;
// __device__ float gamma = 0.9f;
// __device__ float alpha = 0.1f;

// float epsilon = 1.000f;
// float *d_epsilon;


///////////////////////////////    agent_init()    ////////////////////////////// 

__global__ void Agent_init(float *d_epsilon) {
	// works for multiple agents 
	*d_epsilon = 1.000f;
}

__global__ void Qtable_init(float *d_qtable) {  // float *d_qtable
								 // init Q-table Q(s, a) = 0, s in S, a in A(s)
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	d_qtable[tid] = 0;
}

__global__ void Randstate_init(hiprandState *d_randstate) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(clock() + tid, tid, 0, &d_randstate[tid]);
}

void agent_init()
{

	float epsilon = 1.000f;
	// float gamma = 0.900f;
	// float alpha = 0.100f;
	int qSize = NUM_OF_ACTIONS * COLS * ROWS;
	int randSize = COLS * ROWS;

	hipMalloc((void **)&d_qtable, sizeof(float) * qSize);
	hipMalloc((void **)&d_randstate, sizeof(hiprandState) * randSize);

	hipMalloc((void **)&d_action, sizeof(short));
	hipMalloc((void **)&d_action_buff, sizeof(short));
	hipMalloc((void **)&d_epsilon, sizeof(float));

	// hipMemcpy(d_epsilon, &epsilon, sizeof(float), hipMemcpyHostToDevice);
	// all the threads need to maintain their own states 

	Agent_init << <1, 1 >> > (d_epsilon);
	Qtable_init << <NUM_OF_STATES, NUM_OF_ACTIONS >> > (d_qtable); // (d_qtable);
	Randstate_init << <COLS, ROWS >> > (d_randstate);

}

///////////////////////////////    agent_ation()    ////////////////////////////// 

// __global__ void kernel_fun(hiprandState *d_randstate) {
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	hiprand_uniform(&d_randstate[tid]);
// }

__global__ void Agent_action(int2 *cstate, short *d_action, hiprandState *d_randstate, float *d_epsilon, float *d_qtable, short *d_action_buff) {

	int agent_id = 0;
	int x = cstate[agent_id].x;
	int y = cstate[agent_id].y;

	// the x, y coodinate will be block.x block.y in qtable
	// for each pos, there are still 4 action types;
	int action;
	float max_qval;

	// exploration
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float rand_state = hiprand_uniform(&d_randstate[tid]);

	if (rand_state < *d_epsilon) {
		float div = 1.000f / ((float)NUM_OF_ACTIONS);
		action = (int)(rand_state / div);
	}
	else { // exploitation (greedy policy)
		action = 0;
		int qid = (y * COLS + x) * NUM_OF_ACTIONS;
		for (int i = 0; i < NUM_OF_ACTIONS; ++i) {
			if (d_qtable[qid + i] > max_qval) {
				max_qval = d_qtable[qid + i];
				action = i;
			}
		}
	}

	// decide the action
	d_action = (short*)action;
	d_action_buff = d_action;
}


short* agent_action(int2* cstate) {
	// do exploration or exploitation
	Agent_action << <COLS, ROWS >> > (cstate, d_action, d_randstate, d_epsilon, d_qtable, d_action_buff);
	return d_action;
}

/////////////////////////////    agent_update()    ///////////////////////////////

__global__ void Agent_update(int2* cstate, int2* nstate, float *rewards, float *d_qtable, short *d_action_buff) 
{
	// observe next state S' and R
	int agent_id = 0;

	int x0 = cstate[agent_id].x;
	int y0 = cstate[agent_id].y;

	int x1 = nstate[agent_id].x;
	int y1 = nstate[agent_id].y;

	// next state (n+1)
	int n_qid = (y1 * COLS + x1) * NUM_OF_ACTIONS;
	float max_qval = d_qtable[n_qid];
	for (int i = 0; i < NUM_OF_ACTIONS; ++i) {
		if (d_qtable[n_qid + i] > max_qval) {
			max_qval = d_qtable[n_qid + i];
		}
	}

	// update q_table of current state (n) by max val of next state
	// Q(S, A) <- Q(S, A) + alpha[R + gamma * max Q(S', a) - Q(S, A)]
	int c_qid = (y0 * COLS * x0) + *d_action_buff;
	d_qtable[c_qid] += alpha * (rewards[agent_id] + gamma * max_qval - d_qtable[c_qid]);

	// update state to next
	if (rewards[agent_id] == 1 || rewards[agent_id] == -1) {
		// agent status: inactive 
		cstate[agent_id].x = 0;
		cstate[agent_id].y = 0;
	}
	else if (rewards[agent_id] == 0) {
		// agent status: active
		cstate[agent_id] = nstate[agent_id];
	}
}


void agent_update(int2* cstate, int2* nstate, float *rewards)
{
	// add your codes
	Agent_update <<<1, 1 >>> (cstate, nstate, rewards, d_qtable, d_action_buff);
}


/////////////////////////////    adjust_epsilon()    ///////////////////////////////

__global__ void Adjust_epsilon(float *d_epsilon) {
	if (*d_epsilon > 1.000f) {
		*d_epsilon = 1.000f;
	}
	else if (*d_epsilon < 0.100f) {
		*d_epsilon = 0.000f; // not do exploration any more
	}
	else {
		*d_epsilon -= 0.001f;
	}
}


float agent_adjustepsilon()
{
	// add your codes
	Adjust_epsilon << <1, 1 >> > (d_epsilon);
	// hipMemcpy(&epsilon, d_epsilon, sizeof(float), hipMemcpyDeviceToHost);
	float epsilon = *d_epsilon;
	return epsilon;
}
