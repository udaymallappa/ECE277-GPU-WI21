#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// DO NOT change the kernel function
__global__ void vector_add(int *a, int *b, int *c)
{
// DO NOT change the kernel function
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}


#define N (2048*2048)
#define THREADS_PER_BLOCK 128

#define NSTREAMS 4


int main()
{
    int *a, *b, *c, *golden;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	hipHostAlloc((void **)&a, size, hipHostMallocDefault);
	hipHostAlloc((void **)&b, size, hipHostMallocDefault);
	hipHostAlloc((void **)&c, size, hipHostMallocDefault);

	//a = (int *)malloc( size );
	//b = (int *)malloc( size );
	//c = (int *)malloc( size );

	golden = (int *)malloc(size);

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		golden[i] = a[i] + b[i];
		c[i] = 0;
	}

	hipStream_t streams[NSTREAMS];
	for (int i = 0; i < NSTREAMS; ++i) {
		hipStreamCreate(&streams[i]);
	}

	int nsdata = N / NSTREAMS;
	int iBytes = size / NSTREAMS;

	dim3 mgrid((nsdata + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);
	dim3 mblock(THREADS_PER_BLOCK);

	for (int i = 0; i < NSTREAMS; ++i) {
		int offset = i * nsdata;
		
		hipMemcpyAsync(&d_a[offset], &a[offset], iBytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&d_b[offset], &b[offset], iBytes, hipMemcpyHostToDevice, streams[i]);

		vector_add << < mgrid, mblock, 0, streams[i] >> > (&d_a[offset], &d_b[offset], &d_c[offset]);
		hipMemcpyAsync(&c[offset], &d_c[offset], iBytes, hipMemcpyDeviceToHost, streams[i]);
	}

	for (int i = 0; i < NSTREAMS; ++i) {
		hipStreamSynchronize(streams[i]);
	}

	for (int i = 0; i < NSTREAMS; ++i) {
		hipStreamDestroy(streams[i]);
	}

	bool pass = true;
	for (int i = 0; i < N; i++) {
		if (golden[i] != c[i]) {
			pass = false;
			//printf("%i %d %d \n", i, golden[i], c[i]);
		}
	}
	
	if (pass)
		printf("PASS\n");
	else
		printf("FAIL\n");

	printf("print your name and id \n>> Yifan Wang, A53298382 \n\n");

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	//free(a);
	//free(b);
	//free(c);

	free(golden);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 
