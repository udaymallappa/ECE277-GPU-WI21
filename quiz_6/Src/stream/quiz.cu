#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// DO NOT change the kernel function
__global__ void vector_add(int *a, int *b, int *c)
{
// DO NOT change the kernel function
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}


#define N (2048*2048)
#define THREADS_PER_BLOCK 128

#define NSTREAMS 4

int main()
{
    int *a, *b, *c, *golden;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	// GMEM allication gbuf
	hipMalloc( (void **) &d_a, size);
	hipMalloc( (void **) &d_b, size);
	hipMalloc( (void **) &d_c, size);

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	golden = (int *)malloc(size);

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		golden[i] = a[i] + b[i];
		c[i] = 0;
	}

	// hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	// hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	// declared streams
	hipStream_t streams[NSTREAMS];
	int streamSize = size / NSTREAMS; // streamBytes = stream_size * sizeof(float);

	// event start
	// hipEventRecord(addEvent, 0);

	for (int i = 0; i < NSTREAMS; ++i) {

		hipStreamCreate(&streams[i]);

		int offset = i * streamSize;

		// cudaMemocpyAsync(hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&d_a[offset], &a[offset], streamSize, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(&d_b[offset], &b[offset], streamSize, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(&d_c[offset], &c[offset], streamSize, hipMemcpyHostToDevice, stream[i]);

		int mgrid = (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK / NSTREAMS;
		int mblock = THREADS_PER_BLOCK / NSTREAMS;
		vector_add <<< mgrid,  mblock, 0, streams[i]>>>( &d_a[offset], &d_b[offset], &d_c[offset] );
		// cudaMemocpyAsync(hipMemcpyDeviceToHost, streams[i]);
		hipMemcpyAsync(&a[offset], &d_a[offset], streamSize, hipMemcpyDeviceToHost, stream[i]);
		hipMemcpyAsync(&b[offset], &d_b[offset], streamSize, hipMemcpyDeviceToHost, stream[i]);
		hipMemcpyAsync(&c[offset], &d_c[offset], streamSize, hipMemcpyDeviceToHost, stream[i]);

		hipStreamDestroy(streams[i]);

	}

	// event end
	// hipEventRecord(addEvent, 0);

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	bool pass = true;
	for (int i = 0; i < N; i++) {
		if (golden[i] != c[i])
			pass = false;
	}
	
	if (pass)
		printf("PASS\n");
	else
		printf("FAIL\n");

	printf("print your name and id\n");

	free(a);
	free(b);
	free(c);
	free(golden);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 
