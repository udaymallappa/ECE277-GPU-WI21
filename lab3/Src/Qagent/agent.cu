#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 277: GPU Programmming 2021 WINTER quarter
/* Author and Instructer: Cheolhong An
/* Copyright 2019
/* University of California, San Diego
/*************************************************************************/

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// #mines=96, #flag=1

#define COLS	46
#define ROWS	46

#define NUM_OF_AGENTS	512
#define NUM_OF_ACTIONS	4

#define GAMMA		0.9
#define ALPHA		0.1
#define	EPSILON		1.0
#define DELTA_EPS	0.001

short *d_action;
hiprandState *d_state;
bool  *d_active;  
float3 *d_qtable; // .x col, .z row, .z action 

float epsilon;
float *d_epsilon;

/**	Host: 	agent_init()  ////////////////////////////////////////////
* @brief 	clear action + initQ table + self initialization
*/

__global__ void Init_agent(hiprandState *d_state, bool *d_active) {

	unsigned int agent_id = threadIdx.x + blockIdx.x * blockDim.x;
	// d_action[agent_id] = 2;
	hiprand_init(clock() + agent_id, agent_id, 0, &d_state[agent_id]);
	d_active[agent_id] = 1;
}

__global__ void Init_epsilon(float *d_epsilon) {
	*d_epsilon = 1.0f;
}

__global__ void Init_qtable(float *d_qtable) {
	// it's 4 x board_size, as the qtable includes 4 actions at each position.
	// init Q-table Q(s, a) = 0, s in S, a in A(s)
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int tid = iy * (COLS * NUM_OF_ACTIONS) + ix;
	d_qtable[tid] = 0;
}

void agent_init()
{
	// clear action + initQ table + self initialization
	hipMalloc((void **)&d_action, sizeof(short) * NUM_OF_AGENTS);

	hipMalloc((void **)&d_state, sizeof(hiprandState) * NUM_OF_AGENTS);
	hipMalloc((void **)&d_active, sizeof(bool) * NUM_OF_AGENTS);
	Init_agent << <1, NUM_OF_AGENTS >> > (d_state, d_active);

	hipMalloc((void **)&d_epsilon, sizeof(float));
	Init_epsilon << <1, 1 >> > (d_epsilon);

	// int qSize = NUM_OF_ACTIONS * COLS * ROWS;
	hipMalloc((void **)&d_qtable, sizeof(float3) * 1); // *aSize
	// the size is (float3)*1 as there is only one 
	dim3 grid(COLS, ROWS);
	dim3 block(NUM_OF_ACTIONS);
	Init_qtable << <grid, block >> > (d_qtable);

}

/** Host:	agent_init_episode() //////////////////////////////////////////////
* @brief 	set all agents in active status
*/

__global__ void Init_epsiode(bool *d_active) {
	// agent 1 alive, 0 dead;
	unsigned int agent_id = threadIdx.x + blockIdx.x * blockDim.x;
	d_active[agent_id] = 1;
}

void agent_init_episode() {
	// set all agents in active status
	Init_epsiode << <1, NUM_OF_AGENTS >> > (d_active);
}

/** Host: 	adjust_epsilon() ////////////////////////////////////////////
* @brief 	adjust epsilon, return a CPU variable
*/

__global__ void Adjust_epsilon(float *d_epsilon) {
	if (*d_epsilon > 1.0f) {
		*d_epsilon = 1.0f;
	}
	else if (*d_epsilon < 0.0f) {
		*d_epsilon = 0.0f;
	}
	else {
		*d_epsilon -= DELTA_EPS;
	}
	// *d_epsilon -= DELTA_EPS;
}


float agent_adjustepsilon()
{
	Adjust_epsilon << <1, 1 >> > (d_epsilon);
	hipMemcpy(&epsilon, d_epsilon, sizeof(float), hipMemcpyDeviceToHost);
	return epsilon;
}

/** Host:	agent_action() ////////////////////////////////////////////
* @brief	if agent is alive, run algorithm to take action
*/


__global__ void Agent_action(int2 *cstate, short *d_action, hiprandState *d_state, float *d_epsilon, float *d_qtable, bool *d_active) {

	unsigned int agent_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (d_active[agent_id] == 1) {
		// agent is alive 
		unsigned int x = cstate[agent_id].x;
		unsigned int y = cstate[agent_id].y;

		// the x, y coodinate will be block.x block.y in qtable
		// for each pos, there are still 4 action types;

		// exploration
		float rand_state = hiprand_uniform(d_state[agent_id]);
		short action;
		if (rand_state < *d_epsilon) {
			// float div = 1.000f / ((float)NUM_OF_ACTIONS);
			action = (short)(hiprand_uniform(d_state[agent_id]) * NUM_OF_ACTIONS);
			if (action == 4) action = 0;
		}
		else {
			// exploitation (greedy policy)
			int qid = (y * COLS + x) * NUM_OF_ACTIONS;
			// int qid = y * (COLS * NUM_OF_ACTIONS) + (x * NUM_OF_ACTIONS);
			float max_qval = d_qtable[qid];
			action = 0;
			for (unsigned int i = 1; i < NUM_OF_ACTIONS; ++i) {
				if (d_qtable[qid + i] > max_qval) {
					max_qval = d_qtable[qid + i];
					action = (short)i;
				}
			}
		}

		// decide the action
		d_action[agent_id] = action;
	}
}


short* agent_action(int2* cstate) {
	// do exploration or exploitation
	Agent_action << <1, NUM_OF_AGENTS >> > (cstate, d_action, d_state, d_epsilon, d_qtable, d_active); // , d_active
	return d_action;
}

/** Host:	agent_update() ////////////////////////////////////////////
* @brief	if agent is alive, update qtable
*/

__global__ void Agent_update(int2* cstate, int2* nstate, float *rewards, float *d_qtable, short *d_action, bool *d_active)
{
	// observe next state S' and R
	int agent_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (d_active[agent_id] == 1) {
		// agent active
		unsigned int x0 = cstate[agent_id].x;
		unsigned int y0 = cstate[agent_id].y;

		unsigned int x1 = nstate[agent_id].x;
		unsigned int y1 = nstate[agent_id].y;

		float gamma_item = 0;

		if (rewards[agent_id] == 0) {
			// next state (n+1)
			int n_qid = (y1 * COLS + x1) * NUM_OF_ACTIONS;
			// int n_qid = (y1 * COLS * NUM_OF_ACTIONS) + (x1 * NUM_OF_ACTIONS);
			float best_next_qval = d_qtable[n_qid];
			// i start from 1 as the i = 0 has been assign as init max_qval
			for (unsigned int i = 1; i < NUM_OF_ACTIONS; ++i) {
				if (d_qtable[n_qid + i] > best_next_qval) {
					best_next_qval = d_qtable[n_qid + i];
				}
			}
			gamma_item = GAMMA * best_next_qval;
			// agent still active
		}

		// update q_table of current state (n) by max val of next state
		// Q(S, A) <- Q(S, A) + alpha[R + gamma * max Q(S', a) - Q(S, A)]
		unsigned int c_qid = (y0 * COLS + x0) * NUM_OF_ACTIONS + (int)d_action[agent_id];
		// int c_qid = (y0 * COLS * NUM_OF_ACTIONS) + (x0 * NUM_OF_ACTIONS) + (int)d_action[agent_id];
		d_qtable[c_qid] += ALPHA * (rewards[agent_id] + gamma_item - d_qtable[c_qid]);

		// update state to next
		if (rewards[agent_id] == 0) {
			// agent status: active
			cstate[agent_id] = nstate[agent_id];
		}
		// else {
		// 	// agent status: inactive 
		// 	cstate[agent_id].x = 0;
		// 	cstate[agent_id].y = 0;
		// 	d_active[agent_id] = 0;
		// }
	}
}


void agent_update(int2* cstate, int2* nstate, float *rewards)
{
	// add your codes
	Agent_update << <1, NUM_OF_AGENTS >> > (cstate, nstate, rewards, d_qtable, d_action, d_active);
}



