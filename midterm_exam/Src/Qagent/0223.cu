#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 277: GPU Programmming 2021 WINTER quarter
/* Author and Instructer: Cheolhong An
/* Copyright 2019
/* University of California, San Diego
/*************************************************************************/

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define ACTIONS		4
#define NUM_AGENTS	512

#define COLS		46
#define ROWS		46
#define QSIZE 		COLS * ROWS * ACTIONS

#define THREADS		1024
#define WARPSIZE	32

#define GAMMA		0.9
#define ALPHA		0.5
#define	EPSILON		1.0
#define DELTA_EPS	0.01

#define EPS_CEIL	1.0
#define EPS_BOTTOM  0.0


short *d_action;
hiprandState *d_states;

bool  *d_active;
float *d_qtable;

// float2 *d_actval; 	// sizeof() * #agents : .x action, .y qval; 

float epsilon;


//////////////////////////	agent_init()  //////////////////////////

// <<< NUM_AGENTS 512 / WARPSIZE 32 = 16, WARPSIZE 32 = 32 >>>
__global__ void Init_agent(hiprandState *d_states, bool *d_active)
{
	unsigned int agent_id = threadIdx.x + blockIdx.x * blockDim.x;

	hiprand_init(clock() + agent_id, agent_id, 0, &d_states[agent_id]);
	d_active[agent_id] = 1;
}


// <<< (#cols 46/2, #rows 46/2), #actions 4*4 >>> (23, 23), (16)
// occupency
__global__ void Init_qtable(float *d_qtable)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned int nx = gridDim.x * blockDim.x;
	unsigned int tid = ix + iy * nx; 
	d_qtable[tid] = 0.0f;
}


void agent_init()
{
	// clear action + initQ table + self initialization
	epsilon = EPSILON;

	hipMalloc((void **)&d_action, NUM_AGENTS * sizeof(short));
	// hipMalloc((void **)&d_actval, NUM_AGENTS * sizeof(float2)); // used in __device__ void

	hipMalloc((void **)&d_states, NUM_AGENTS * sizeof(hiprandState));
	hipMalloc((void **)&d_active, NUM_AGENTS * sizeof(bool));

	dim3 block(WARPSIZE, 1, 1);
	dim3 grid(NUM_AGENTS / WARPSIZE, 1, 1);
	// dim3 block(512, 1, 1), dim3 grid(1, 1, 1);
	Init_agent << <grid, block >> > (d_states, d_active);

	hipMalloc((void **)&d_qtable, QSIZE * sizeof(float));

	dim3 qblock(ACTIONS * 4, 1, 1);
	dim3 qgrid(COLS / 2, ROWS / 2, 1);
	// dim3 qblock(4, 1, 1), dim3 qgrid(46, 46, 1);
	Init_qtable << < qgrid, qblock >> > (d_qtable);		// <<< (23, 23), 16 >>>

}


//////////////////////////	agent_init_episode() //////////////////////////

// <<< NUM_AGENTS 512 / WARPSIZE = 16, WARPSIZE = 32 >>>
__global__ void Init_epsiode(bool *d_active) {
	// agent 1 alive, 0 dead;
	unsigned int agent_id = threadIdx.x + blockIdx.x * blockDim.x;
	d_active[agent_id] = 1;
}


void agent_init_episode() {
	// set all agents in active status
	dim3 block(WARPSIZE, 1, 1);
	dim3 grid(NUM_AGENTS / WARPSIZE, 1, 1);
	// dim3 block(512, 1, 1);
	// dim3 grid(1, 1, 1);
	Init_epsiode << <grid, block >> > (d_active);
}


//////////////////////////	adjust_epsilon() //////////////////////////


float agent_adjustepsilon()
{
	if (epsilon > EPS_CEIL) {
		epsilon = EPS_CEIL;
	}
	else if (epsilon < EPS_BOTTOM) {
		epsilon = EPS_BOTTOM;
	}
	else {
		epsilon -= DELTA_EPS;
	}
	return epsilon;
}


//////////////////////////	agent_action() //////////////////////////




// <<< NUM_AGENTS 512 * ACTIONS 4 / THREADS = 2 , THREADS = 1024 >>>
__global__ void Agent_action(int2 *cstate, short *d_action, hiprandState *d_states, float epsilon, float *d_qtable, bool *d_active)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int agent_id = tid / ACTIONS;
	// unsigned int agent_id = blockIdx.x; 

	if (d_active[agent_id] == 1)
	{
		// agent is alive 
		float rand_state = hiprand_uniform(&d_states[agent_id]);

		if (rand_state < epsilon) {
			// exploration
			short action = (short)(hiprand_uniform(&d_states[agent_id]) * ACTIONS);
			if (action == 4) {
				// hiprand_uniform (0, 1] for keeping uniform make the case action==4 as action==0
				action = 0;
			}
			d_action[agent_id] = action;
		}
		else {
			// exploitation (greedy policy)
			// Get_qAction_qMaxVal << <1, ACTIONS >> > (cstate, d_qtable, d_actval, agent_id);

			unsigned int x = cstate[agent_id].x;
			unsigned int y = cstate[agent_id].y;

			unsigned int sid = threadIdx.x;  	// 0123 4567 .. ...1023

			// extern __shared__ float qval_cache[]; 
			// extern __shared__ short action_cache[];

			__shared__ float qval_cache[THREADS]; 
			__shared__ short action_cache[THREADS];

			// unsigned int aid = tid - agent_id * ACTIONS;
			unsigned int aid = sid % 4;
			action_cache[sid] = (short)aid;  // 0123 0123 .. ...3 

			unsigned int q_id = (y * COLS + x) * ACTIONS;
			qval_cache[sid] = d_qtable[q_id + aid];  // 

			__syncthreads();

			unsigned int stride = ACTIONS / 2; 

			#pragma unroll
			while (stride != 0) {
				// 1st round, stride==2 : 01(23), 0<-2  1<-3
				// 2nd round, stride==1 : 0(123), 0<-1
				if (aid < stride && qval_cache[sid] < qval_cache[sid + stride]) {
					// keep larger values in left cache
					qval_cache[sid] = qval_cache[sid + stride];
					action_cache[sid] = action_cache[sid + stride];
				}
				__syncthreads();
				stride /= 2;
			}
			// update: .x action; .y max_qval.
			// d_actval[agent_id].x = action_cache[agent_id * ACTIONS];
			// d_actval[agent_id].y = (float)qval_cache[agent_id * ACTIONS];
			// d_action[agent_id] = action_cache[agent_id * ACTIONS];
			d_action[agent_id] = action_cache[sid / ACTIONS];
			// d_action[agent_id] = (short)d_actval[agent_id].x; // .x action .y max_qval
		}
		// decide the action
		// d_action[agent_id] = action;
	}
}


short* agent_action(int2* cstate) {
	// do exploration or exploitation
	dim3 block(THREADS, 1, 1);
	dim3 grid(NUM_AGENTS * ACTIONS / THREADS, 1, 1);
	// dim3 block(4, 1, 1);
	// dim3 grid(512, 1, 1);
	// <<< NUM_AGENTS 512 * ACTIONS 4 / THREADS = 2 , THREADS = 1024 >>>
	Agent_action <<< grid, block >>> (cstate, d_action, d_states, epsilon, d_qtable, d_active);
	return d_action;
}


//////////////////////////	agent_update() //////////////////////////

// <<< NUM_AGENTS 512 * ACTIONS 4 / THREADS = 2 , THREADS = 1024 >>>
__global__ void Agent_update(int2* cstate, int2* nstate, float *rewards, float *d_qtable, short *d_action, bool *d_active)
{
	// observe next state S' and R
	// unsigned int agent_id = threadIdx.y;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int agent_id = tid / ACTIONS;
	unsigned int agent_id = blockIdx.x;

	if (d_active[agent_id] == 1) {
		// agent active
		float gamma_item = 0; // if agent is inactive, the gamma_item == 0

		if (rewards[agent_id] == 0) {
			// agent still active
			// Get_qAction_qMaxVal << < 1, ACTIONS >> > (nstate, d_qtable, d_actval, agent_id);

			unsigned int x = nstate[agent_id].x;
			unsigned int y = nstate[agent_id].y;		

			// extern __shared__ float qval_cache[]; 
			// extern __shared__ short action_cache[];

			__shared__ float qval_cache[4]; 
			__shared__ short action_cache[4];

			unsigned int aid = tid - agent_id * ACTIONS;
			action_cache[aid] = aid;  // 0123 .. 0123 0123 ..

			unsigned int q_id = (y * COLS + x) * ACTIONS;
			qval_cache[aid] = d_qtable[q_id + aid];  // 

			__syncthreads();

			unsigned int stride = ACTIONS / 2; 

			#pragma unroll
			while (stride != 0) {
				// 1st round, stride==2 : 01(23), 0<-2  1<-3
				// 2nd round, stride==1 : 0(123), 0<-1
				if (aid < stride && qval_cache[aid] < qval_cache[aid + stride]) {
					// keep larger values in left cache
					qval_cache[aid] = qval_cache[aid + stride];
					action_cache[aid] = action_cache[aid + stride];
				}
				__syncthreads();
				stride /= 2;
			}
			// update: .x action; .y max_qval.
			// d_actval[agent_id].x = action_cache[agent_id * ACTIONS];
			// d_actval[agent_id].y = qval_cache[agent_id * ACTIONS];

			// float best_next_qval = d_actval[agent_id].y; // .x action .y max_qval
			float best_next_qval = qval_cache[0];
			gamma_item = GAMMA * best_next_qval; //qval_cache[agent_id * ACTIONS]; // max qval for next state
		}

		// update q_table of current state (n) <- max val of next state (n+1)
		// Q(S, A) <- Q(S, A) + alpha[R + gamma * max Q(S', a) - Q(S, A)]

		unsigned int x0 = cstate[agent_id].x;
		unsigned int y0 = cstate[agent_id].y;

		unsigned int c_qid = (y0 * COLS + x0) * ACTIONS + (unsigned int)d_action[agent_id];
		d_qtable[c_qid] += ALPHA * (rewards[agent_id] + gamma_item - d_qtable[c_qid]);
	}
}


void agent_update(int2* cstate, int2* nstate, float *rewards)
{
	// dim3 block(THREADS, 1, 1);
	// dim3 grid(NUM_AGENTS * ACTIONS / THREADS, 1, 1);
	dim3 block(4, 1, 1);
	dim3 grid(512, 1, 1);
	// <<< NUM_AGENTS 512 * ACTIONS 4 / THREADS = 2 , THREADS = 1024 >>>
	Agent_update <<< grid, block >>> (cstate, nstate, rewards, d_qtable, d_action, d_active);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////


/** <<< 1, #actions 4 >>>  CUDA Dynamic Parallelism
* @brief it will be called in __global__ Agent_action and Agent_update
* 		  for agent_id to calculate (.x) greedy_action and (.y) max_qval
* @param cstate 	int2
* @param d_qtable 	float
* @param d_actval 	float2
* @param agent_id 	unsigned int
* @return __device__ void
*/

// __inline__ __device__ void Get_qAction_qMaxVal(int2 *state, float *d_qtable, float2 *d_actval, unsigned int agent_id)
// {
// 	// exploitation (greedy policy)

// 	// located position on q_table
// 	unsigned int x = state[agent_id].x;
// 	unsigned int y = state[agent_id].y;

// 	// memory shared
// 	__shared__ float qval_cache[ACTIONS]; // 4 actions  
// 	__shared__ short action_cache[ACTIONS];

// 	unsigned int aid = threadIdx.x; // action_id
// 	action_cache[aid] = (short)threadIdx.x;

// 	unsigned int q_id = (y * COLS + x) * ACTIONS;
// 	qval_cache[aid] = d_qtable[q_id + aid];

// 	__syncthreads();

// 	// reduction for getting the max val and action
// 	unsigned int stride = blockDim.x / 2; // 4 actions / 2

// 	#pragma unroll
// 	while (stride != 0) {
// 		if (aid < stride && qval_cache[aid] < qval_cache[aid + stride]) {
// 			// keep larger values in left cache
// 			qval_cache[aid] = qval_cache[aid + stride];
// 			action_cache[aid] = action_cache[aid + stride];
// 		}
// 		__syncthreads();
// 		stride /= 2;
// 	}
// 	// update: .x action; .y max_qval.
// 	d_actval[agent_id].x = action_cache[0];
// 	d_actval[agent_id].y = qval_cache[0];
// }